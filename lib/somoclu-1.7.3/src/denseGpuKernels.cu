#include "hip/hip_runtime.h"
/**
 * Self-Organizing Maps on a cluster
 *  Copyright (C) 2013 Peter Wittek
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#undef _GLIBCXX_ATOMIC_BUILTINS
#undef _GLIBCXX_USE_INT128

#include <iostream>
#include <sstream>
#include <map>
#include <vector>
#include <stdio.h>
#include <hipblas.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>

#include "somoclu.h"

#ifdef _WIN32
#define popen _popen
#define pclose _pclose
#endif

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        stringstream sstm; \
        sstm << "CUDA error calling \""#call"\", code is " << err; \
        my_abort(sstm.str()); }

//Globals
hipblasHandle_t handle;
thrust::device_vector<float> deviceData;
thrust::device_vector<float> deviceDataNorms;
thrust::device_vector<float> deviceCodebook;
thrust::device_vector<float> deviceCodebookNorms;

// convert a linear index to a row index
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T, T> {
    T C; // number of columns

    __host__ __device__
    linear_index_to_row_index(T C) : C(C) {}

    __host__ __device__
    T operator()(T i) {
        return i / C;
    }
};

// note: functor inherits from unary_function
template <typename T>
struct square : public thrust::unary_function<T, T> {
    __host__ __device__
    T operator()(T x) const {
        return x * x;
    }
};

typedef thrust::tuple<int, float> argMinType;

struct argMin : public thrust::binary_function<argMinType, argMinType, argMinType> {
    __host__ __device__
    argMinType operator()(const argMinType& a, const argMinType& b) const {
        if (thrust::get<1>(a) < thrust::get<1>(b)) {
            return a;
        }
        else {
            return b;
        }
    }
};

template <typename T>
thrust::device_vector<T> normsOfRowSpace(thrust::device_vector<T> A, int nRows, int nColumns) {
    // allocate storage for row sums and indices
    thrust::device_vector<T> row_sums(nRows);
    thrust::device_vector<int> row_indices(nRows);

    // compute row sums by summing values with equal row indices
    thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(nColumns)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(nColumns)) + (nRows * nColumns),
     thrust::make_transform_iterator(A.begin(), square<T>()),
     row_indices.begin(),
     row_sums.begin(),
     thrust::equal_to<int>(),
     thrust::plus<T>());

    return row_sums;
}

thrust::device_vector<argMinType> minsOfRowSpace(thrust::device_vector<float> A, int nRows, int nColumns) {
    // allocate storage for row sums and indices
    thrust::device_vector<argMinType> row_sums(nRows);
    thrust::device_vector<int> row_indices(nRows);

    // compute row sums by summing values with equal row indices
    thrust::reduce_by_key
    (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(nColumns)),
     thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(nColumns)) + (nRows * nColumns),
     thrust::make_zip_iterator(thrust::make_tuple(thrust::counting_iterator<int>(0), A.begin())),
     row_indices.begin(),
     row_sums.begin(),
     thrust::equal_to<int>(),
     argMin());
    return row_sums;
}

template <int BLOCK_DIM>
__global__ void euclidean(float *anorm2, float *bnorm2, float *M, int height, int width) {
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yStartIndex = blockIdx.y * BLOCK_DIM;
    if (xIndex < width) {
        float bNormForX = bnorm2[xIndex];
        unsigned int yEndIndex = (yStartIndex + BLOCK_DIM < height ? yStartIndex + BLOCK_DIM : height);
        for (unsigned int yIndex = yStartIndex; yIndex < yEndIndex; yIndex++) {
            unsigned int index = yIndex * width + xIndex;
            M[index] = anorm2[yIndex] - 2 * M[index] + bNormForX;
        }
    }
}

template <typename T>
void printMatrix(thrust::device_vector<T> A, int nRows, int nColumns) {
    for (size_t i = 0; i < nRows; i++) {
        for (size_t j = 0; j < nColumns; j++) {
            std::cout << A[i * nColumns + j] << " ";
        }
        std::cout << "\n";
    }
    std::cout << "\n";
}

/** Clear the device memory and shut down CUBLAS
 *
 */
void freeGpu() {
    deviceData.clear();
    deviceDataNorms.clear();
    deviceCodebook.clear();
    deviceCodebookNorms.clear();
    thrust::device_vector<float>().swap(deviceData);
    thrust::device_vector<float>().swap(deviceDataNorms);
    thrust::device_vector<float>().swap(deviceCodebook);
    thrust::device_vector<float>().swap(deviceCodebookNorms);
    hipblasStatus_t status = hipblasDestroy(handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        my_abort("CuBLAS shutdown error");
    }
}

/** Find the best matching units -- called from the map function
 * @param bmus - array of best matching units
 * @param codebook - the codebook to save
 * @param nSomX - dimensions of SOM map in the x direction
 * @param nSomY - dimensions of SOM map in the y direction
 * @param nDimensions - dimensions of a data instance
 * @param nVectorsPerRank - the number of data points assigned to this GPU
 */

void getBmusOnGpu(int *bmus, float *codebook, int nSomX, int nSomY, int nDimensions, int nVectorsPerRank) {
    deviceCodebook = thrust::device_vector<float>(codebook, codebook + nSomX * nSomY * nDimensions);
    deviceCodebookNorms = normsOfRowSpace<float>(deviceCodebook, nSomX * nSomY, nDimensions);
    thrust::device_vector<float> deviceGramMatrix(nSomX * nSomY * nVectorsPerRank, 0);
    //Calculate the inner products of the data vectors and the weight vectors

    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasStatus_t status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                        nSomX * nSomY, nVectorsPerRank, nDimensions,
                                        &alpha, thrust::raw_pointer_cast(&deviceCodebook[0]), nDimensions,
                                        thrust::raw_pointer_cast(&deviceData[0]), nDimensions,
                                        &beta,  thrust::raw_pointer_cast(&deviceGramMatrix[0]), nSomX * nSomY);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        my_abort("Kernel execution error.");
    }

    //All components of the vectorized Euclidean distance are available
    // 32 is a magic number, this is the block size that works best on Tesla C2050
    int BLOCK_DIM = 32;
    dim3 grid((nSomX * nSomY + BLOCK_DIM - 1) / BLOCK_DIM, (nVectorsPerRank + BLOCK_DIM - 1) / BLOCK_DIM, 1);
    dim3 threads(BLOCK_DIM, 1, 1);
    if (BLOCK_DIM == 32) {
        euclidean<32> <<< grid, threads>>>(thrust::raw_pointer_cast(&deviceDataNorms[0]),
                                           thrust::raw_pointer_cast(&deviceCodebookNorms[0]),
                                           thrust::raw_pointer_cast(&deviceGramMatrix[0]),
                                           nVectorsPerRank, nSomX * nSomY);
    }
    //Finding minimums
    thrust::host_vector<argMinType> minsOfA = minsOfRowSpace(deviceGramMatrix, nVectorsPerRank, nSomX * nSomY);
    CUDA_CHECK(hipDeviceSynchronize());

    //Getting back SOM coordinates from minimums
    for (int i = 0; i < nVectorsPerRank; i++) {
        argMinType tmp = minsOfA[i];
        int somCoordinate = thrust::get<0>(tmp) % (nSomX * nSomY);
        bmus[i * 2] = somCoordinate % nSomX;
        bmus[i * 2 + 1] = somCoordinate / nSomX;
    }
}

/** Initialize CUBLAS and device data
 * @param hostData - the data in the main memory
 * @param height - number of data points assigned to this GPU
 * @param width - dimensions of a data instance
 */

void initializeGpu(float *hostData, int nVectorsPerRank, int nDimensions, int nSomX, int nSomY) {
    /* Initialize CUBLAS */
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        my_abort("CuBLAS initialization error");
    }
    deviceData = thrust::device_vector<float>(hostData, hostData + nVectorsPerRank * nDimensions);
    deviceDataNorms = normsOfRowSpace<float>(deviceData, nVectorsPerRank, nDimensions);
    deviceCodebook = thrust::device_vector<float>(nSomX * nSomY * nDimensions, 0);
    deviceCodebookNorms = thrust::device_vector<float>(nSomX * nSomY, 0);
}

/** Check and initialize a device attached to a node
 * @param commRank - the MPI rank of this process
 * @param commSize - the size of MPI comm world
 */

/// Note that this function was lifted from http://code.google.com/p/gpmr/
void setDevice(int commRank, int commSize) {
    int devCount;
    int deviceNum = 0;
    CUDA_CHECK(hipGetDeviceCount(&devCount));
#ifdef HAVE_MPI
#ifdef _WIN32
    FILE * fp = popen("hostname.exe", "r");
#else
    FILE * fp = popen("/bin/hostname", "r");
#endif
    char buf[1024];
    if (fgets(buf, 1023, fp) == NULL) strcpy(buf, "localhost");
    pclose(fp);
    string host = buf;
    host = host.substr(0, host.size() - 1);
    strcpy(buf, host.c_str());
    if (commRank == 0) {
        map<string, vector<int> > hosts;
        map<string, int> devCounts;
        hosts[buf].push_back(0);
        devCounts[buf] = devCount;

        MPI_Status stat;
        MPI_Request req;
        for (int i = 1; i < commSize; ++i) {
            MPI_Recv(buf, 1024, MPI_CHAR, i, 0, MPI_COMM_WORLD, &stat);
            MPI_Recv(&devCount, 1, MPI_INT, i, 0, MPI_COMM_WORLD, &stat);

            // check to make sure each process on each node reports the same number of devices.
            hosts[buf].push_back(i);
            if (devCounts.find(buf) != devCounts.end()) {
                if (devCounts[buf] != devCount) {
                    printf("Error, device count mismatch %d != %d on %s\n", devCounts[buf], devCount, buf);
                    fflush(stdout);
                }
            }
            else devCounts[buf] = devCount;
        }
        // check to make sure that we don't have more jobs on a node than we have GPUs.
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            if (it->second.size() > static_cast<unsigned int>(devCounts[it->first])) {
                stringstream sstm;
                sstm << "Error, more jobs running on " << it->first.c_str() << " than devices - " << static_cast<int>(it->second.size()) << " jobs > " << devCounts[it->first] << " devices.";
                my_abort(sstm.str());
            }
        }

        // send out the device number for each process to use.
        MPI_Irecv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &req);
        for (map<string, vector<int> >::iterator it = hosts.begin(); it != hosts.end(); ++it) {
            for (unsigned int i = 0; i < it->second.size(); ++i) {
                int devID = i;
                MPI_Send(&devID, 1, MPI_INT, it->second[i], 0, MPI_COMM_WORLD);
            }
        }
        MPI_Wait(&req, &stat);
    }
    else {
        // send out the hostname and device count for your local node, then get back the device number you should use.
        MPI_Status stat;
        MPI_Send(buf, strlen(buf) + 1, MPI_CHAR, 0, 0, MPI_COMM_WORLD);
        MPI_Send(&devCount, 1, MPI_INT, 0, 0, MPI_COMM_WORLD);
        MPI_Recv(&deviceNum, 1, MPI_INT, 0, 0, MPI_COMM_WORLD, &stat);
    }
    MPI_Barrier(MPI_COMM_WORLD);
#endif
    CUDA_CHECK(hipSetDevice(deviceNum));
}

/** One epoch on the GPU, dense variant
 */
void trainOneEpochDenseGPU(int itask, float *data, float *numerator,
                           float *denominator, float *codebook,
                           unsigned int nSomX, unsigned int nSomY,
                           unsigned int nDimensions, unsigned int nVectors,
                           unsigned int nVectorsPerRank, float radius,
                           float scale, string mapType, string gridType,
                           bool compact_support, bool gaussian,
                           int *globalBmus, bool only_bmus, float std_coeff) {
    int *bmus;
#ifdef HAVE_MPI
    bmus = new int[nVectorsPerRank * 2];
#else
    bmus = globalBmus;
#endif
    getBmusOnGpu(bmus, codebook, nSomX, nSomY, nDimensions, nVectorsPerRank);
    if (only_bmus) {
#ifdef HAVE_MPI
        MPI_Gather(bmus, nVectorsPerRank * 2, MPI_INT, globalBmus, nVectorsPerRank * 2, MPI_INT, 0, MPI_COMM_WORLD);
        delete [] bmus;
#endif
        return;
    }
#ifdef HAVE_MPI
    float *localNumerator = new float[nSomY * nSomX * nDimensions];
    float *localDenominator = new float[nSomY * nSomX];
    #pragma omp for
#ifdef _WIN32
    for (int som_y = 0; som_y < nSomY; som_y++) {
#else
    for (unsigned int som_y = 0; som_y < nSomY; som_y++) {
#endif // _WIN32
        for (unsigned int som_x = 0; som_x < nSomX; som_x++) {
            localDenominator[som_y * nSomX + som_x] = 0.0;
            for (unsigned int d = 0; d < nDimensions; d++)
                localNumerator[som_y * nSomX * nDimensions + som_x * nDimensions + d] = 0.0;
        }
    }
    #pragma omp parallel default(shared)
#else  // not HAVE_MPI
    float *localNumerator;
    float localDenominator = 0;
    #pragma omp parallel default(shared) private(localDenominator) private(localNumerator)
#endif
    {
#ifndef HAVE_MPI
        localNumerator = new float[nDimensions];
#endif // HAVE_MPI
        #pragma omp for
#ifdef _WIN32
        for (int som_y = 0; som_y < nSomY; som_y++) {
#else
        for (unsigned int som_y = 0; som_y < nSomY; som_y++) {
#endif
            for (unsigned int som_x = 0; som_x < nSomX; som_x++) {
                for (unsigned int n = 0; n < nVectorsPerRank; n++) {
                    if (itask * nVectorsPerRank + n < nVectors) {
                        float dist = 0.0f;
                        if (gridType == "rectangular") {
                            if (mapType == "planar") {
                                dist = euclideanDistanceOnPlanarMap(som_x, som_y, bmus[2 * n], bmus[2 * n + 1]);
                            }
                            else if (mapType == "toroid") {
                                dist = euclideanDistanceOnToroidMap(som_x, som_y, bmus[2 * n], bmus[2 * n + 1], nSomX, nSomY);
                            }
                        }
                        else {
                            if (mapType == "planar") {
                                dist = euclideanDistanceOnHexagonalPlanarMap(som_x, som_y, bmus[2 * n], bmus[2 * n + 1]);
                            }
                            else if (mapType == "toroid") {
                                dist = euclideanDistanceOnHexagonalToroidMap(som_x, som_y, bmus[2 * n], bmus[2 * n + 1], nSomX, nSomY);
                            }
                        }
                        float neighbor_fuct = getWeight(dist, radius, scale, compact_support, gaussian, std_coeff);
#ifdef HAVE_MPI
                        for (unsigned int d = 0; d < nDimensions; d++) {
                            localNumerator[som_y * nSomX * nDimensions + som_x * nDimensions + d] +=
                                1.0f * neighbor_fuct
                                * (*(data + n * nDimensions + d));
                        }
                        localDenominator[som_y * nSomX + som_x] += neighbor_fuct;
#else // In this case, we can update in place
                        if (n == 0) {
                            localDenominator = neighbor_fuct;
                            for (unsigned int d = 0; d < nDimensions; d++) {
                                localNumerator[d] = 1.0f * neighbor_fuct
                                    * (*(data + n * nDimensions + d));
                            }
                         } else {
                            localDenominator += neighbor_fuct;
                            for (unsigned int d = 0; d < nDimensions; d++) {
                                localNumerator[d] += 1.0f * neighbor_fuct
                                    * (*(data + n * nDimensions + d));
                            }
                         }
#endif // HAVE_MPI                        
                    }
                } // Looping over data instances
#ifndef HAVE_MPI // We update in-place
                for (unsigned int d = 0; d < nDimensions; d++) {
                  if (localDenominator != 0) {
                    float newWeight = localNumerator[d] / localDenominator;
                    codebook[som_y * nSomX * nDimensions + som_x * nDimensions + d] = newWeight;
                  }
                }
#endif
            } // Looping over som_x
        } // Looping over som_y
#ifndef HAVE_MPI
    delete [] localNumerator;
#endif
    } // OPENMP
#ifdef HAVE_MPI
    MPI_Reduce(localNumerator, numerator,
               nSomY * nSomX * nDimensions, MPI_FLOAT, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Reduce(localDenominator, denominator,
               nSomY * nSomX, MPI_FLOAT, MPI_SUM, 0, MPI_COMM_WORLD);
    MPI_Gather(bmus, nVectorsPerRank * 2, MPI_INT, globalBmus, nVectorsPerRank * 2, MPI_INT, 0, MPI_COMM_WORLD);
    delete [] bmus;
    delete [] localNumerator;
    delete [] localDenominator;
#endif
}
